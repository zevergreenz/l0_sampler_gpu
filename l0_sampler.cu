#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <fcntl.h>

#define BUFFER_SIZE 64
#define P 179426239 // a large prime number
#define Z 2       // a random number from [0,P-1]

// X terms produces an x-independent hash
#define NUMCOEFF 4

#define K 15

typedef struct
{
  unsigned int sum;
  unsigned int weight;
  unsigned int fingerprint;
} one_sparse_sampler;

typedef struct
{
  // Number of hash functions
  unsigned int k;

  // Number of one-sparse samplers in each row
  unsigned int s;

  // Number of coefficients in the hash function
  unsigned int numCoefficients;

  // coefficients
  unsigned int *coefficients;
  one_sparse_sampler *samplers;
} s_sparse_sampler;

typedef struct {
  unsigned int n;
  s_sparse_sampler *samplers;
} l0_sampler;

long long wall_clock_time()
{
#ifdef __linux__
	struct timespec tp;
	clock_gettime(CLOCK_REALTIME, &tp);
	return (long long)(tp.tv_nsec + (long long)tp.tv_sec * 1000000000ll);
#else
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (long long)(tv.tv_usec * 1000 + (long long)tv.tv_sec * 1000000000ll);
#endif
}

void checkCudaError(hipError_t rc)
{
  if (rc != hipSuccess)
    printf("Last CUDA error %s\n", hipGetErrorString(rc));
}
void checkCudaError(){
  hipError_t rc = hipGetLastError();
  checkCudaError(rc);
}
unsigned int hash(unsigned int *coeff, unsigned int numCoefficients, unsigned int value) {
  unsigned int val = value;

  for (unsigned int i = 0; i < numCoefficients; i++) {
    val = (val * coeff[i] + val) % P;
  }
  return val;
}

__device__ unsigned int hash_gpu(unsigned int *coeff, unsigned int numCoefficients, unsigned int value) {
  unsigned int val = value;

  for (unsigned int i = 0; i < numCoefficients; i++) {
    val = (val * coeff[i] + val) % P;
  }
  return val;
}

void process_s_sparse(s_sparse_sampler sampler, unsigned int *buffer) {
  unsigned int index, update, hashVal;
  one_sparse_sampler *one_sampler;

  for (unsigned int j = 0; j < BUFFER_SIZE; j+= 2) {
    for (unsigned int i = 0; i < sampler.k; i++) {
      index   = buffer[j];
      update  = buffer[j + 1];
      hashVal = hash(&(sampler.coefficients[i * sampler.numCoefficients]),
                     sampler.numCoefficients,
                     index) % (2 * sampler.s);
      if(index)
        // printf("Updating Sampler (%i, %i) with value %u %u\n", i, hashVal, index, update);
      one_sampler               = &sampler.samplers[i * 2 * sampler.s + hashVal];
      one_sampler->weight      += update;
      one_sampler->sum         += index * update;
      one_sampler->fingerprint += (update * pow(Z, index));
      one_sampler->fingerprint %= P;
    }
  }
}
void process(l0_sampler sampler, unsigned int *buffer) {
  for(int i = 0; i < sampler.n; i++) {
    process_s_sparse(sampler.samplers[i], buffer);
  }
}

__device__ unsigned int powMod(unsigned int z, unsigned int index) {
  unsigned int subpow;
  if(index == 0) {
    return 1;
  } else if(index & 1 == 1) {
    subpow = powMod(z, index >> 1);
    return (z * subpow * subpow) % P;
  } else {
    subpow = powMod(z, index >> 1);
    return (subpow * subpow) % P;
  }
}

__global__ void process_gpu(l0_sampler l0_sampler, unsigned int *buffer) {
  unsigned int i, j, index, update, hashVal, m, s, k;
  s_sparse_sampler *sampler;
  one_sparse_sampler *one_sampler;
  i = threadIdx.x;
  j = threadIdx.y;
  k = blockIdx.x;

  sampler = &(l0_sampler.samplers[j]);

  m = sampler->numCoefficients;
  s = sampler->s;

  index = buffer[k * 2];
  update = buffer[1 + (k * 2)];
  hashVal = hash_gpu( &(sampler->coefficients[i * m]),
           m,
           index) % (2 * sampler->s);
  one_sampler               = &sampler->samplers[i * 2 * s + hashVal];
  // printf("I'm at %i\n", i);
  atomicAdd(&(one_sampler->weight), update);
  atomicAdd(&(one_sampler->sum), index *update);
  // This is slow! pow() only exists for single and double-precision floats
  // And we need a double to cover the range of unsigned integers
  atomicAdd(&(one_sampler->fingerprint), (update * powMod(Z, index) % P));
}

/**
   This method returns an array containing all non-zero indices in the s_sparse
      sampler.
   As of now, this method may contain duplicated indices.
   TODO: Remove duplicates.
 */
unsigned int* query(s_sparse_sampler sampler, unsigned int& size) {
  unsigned int *result = (unsigned int *)malloc(2 * sampler.s * sampler.k * sizeof(unsigned int));

  hipDeviceSynchronize();
  size = 0;
  one_sparse_sampler *one_sampler;

  for (int i = 0; i < sampler.k; i++) {
    for (int j = 0; j < sampler.s * 2; j++) {
      one_sampler = &(sampler.samplers[i * 2 * sampler.s + j]);
      // printf("sampler (%i, %i) has weight %u\n", i, j, one_sampler->weight);
      if (one_sampler->weight != 0) {
        unsigned int index = one_sampler->sum / one_sampler->weight;
        unsigned int error = one_sampler->fingerprint -
                    ((one_sampler->weight * pow(Z, index)));
        if (error % P == 0) result[size++] = index;
      }
    }
  }
  return result;
}

void initialize_s_sparse_sampler(s_sparse_sampler *sampler,
                                 unsigned int               s,
                                 unsigned int               k,
                                 unsigned int               n) {
  int i = 0;
  sampler->k               = k;
  sampler->s               = s;
  sampler->numCoefficients = n;
  hipMallocManaged((void **)&(sampler->samplers),     sizeof(one_sparse_sampler) * k * s * 2);
  hipMallocManaged((void **)&(sampler->coefficients), sizeof(unsigned int) * k * n);

  for(i = 0; i < k * s * 2; i++) {
    sampler->samplers[i].sum = 0;
    sampler->samplers[i].weight = 0;
    sampler->samplers[i].fingerprint = 0;
  }
  for (i = 0; i < k * n; i++)
  {
    sampler->coefficients[i] = rand();
  }
}
void initialize_l0_sampler(l0_sampler *sampler, int s, int k, int m, int n) {
  int i = 0;
  sampler->n = n;
  hipMallocManaged((void **)&(sampler->samplers), sizeof(s_sparse_sampler) * n);
  for (i = 0; i < n; i++) {
    initialize_s_sparse_sampler(&(sampler->samplers[i]), s, k, m);
  }
}

void sample(char *filename, unsigned int s, unsigned int k, unsigned int n) {
  l0_sampler seq_sampler, gpu_sampler;
  
  unsigned int *buffer, *buffer2;
  hipError_t rc;

  long long start_time;
  float seq_time = 0, gpu_time = 0;

  initialize_l0_sampler(&seq_sampler, s, k, NUMCOEFF, n);
  initialize_l0_sampler(&gpu_sampler, s, k, NUMCOEFF, n);

  rc = hipMallocManaged((void **)&buffer, sizeof(unsigned int) * BUFFER_SIZE);
  checkCudaError(rc);

  int i;


  // First evaluate sequential program, then parallel program
  // Read data from file
  FILE *fdIn = fopen(filename, "r");
  start_time = wall_clock_time();
  while (!feof(fdIn))
  {
    for(i = 0; i < BUFFER_SIZE; i+= 2) {
      if(feof(fdIn)) {
        buffer[i] = 0;
        buffer[i + 1] = 0;
      } else {
        fscanf(fdIn, "%u %u", &buffer[i], &buffer[i + 1]);
      }
    }
    process(seq_sampler, buffer);

  }
  seq_time = (float)(wall_clock_time() - start_time) / 1000000000;
  fclose(fdIn);

  unsigned int size;
  unsigned int *result;
  printf("Sequential Vector: Time %1.2f s\n", seq_time);
  // Query the s-sparse sampler and print out
  size = 0;
  // result = query(seq_sampler, size);

  // for (i = 0; i < size; i++)
  //   printf("%u ", result[i]);
  // printf("\n");


  // Read data from file
  fdIn = fopen(filename, "r");
  dim3 blocks(k, n);
  int flip_flop = 0;
  unsigned int *readBuffer = buffer;
  start_time = wall_clock_time();
  while (!feof(fdIn))
  {
    hipDeviceSynchronize();
    // Basically, process one buffer while reading in the next one
    // if(flip_flop) {
    //   readBuffer = buffer;
    // } else {
    //   readBuffer = buffer2;
    // }
    for (i = 0; i < BUFFER_SIZE; i+=2)
    {
      if (feof(fdIn))
      {
        buffer[i] = 0;
        buffer[i + 1] = 0;
      }
      else
      {
        // printf("Reading values for: %i,%i to %#10X \n", i, i + 1, &readBuffer[i]);
        fscanf(fdIn, "%u %u", &buffer[i], &buffer[i + 1]);
        // printf("Read values for:%i : %i, %i\n", i, readBuffer[i], readBuffer[i + 1]);
      }
    }
    // Synchronize parallel blocks.
    process_gpu<<<BUFFER_SIZE / 2, blocks>>>(gpu_sampler, buffer);
    checkCudaError();
    flip_flop = 1 - flip_flop;
  }
  hipDeviceSynchronize();
  gpu_time = (float)(wall_clock_time() - start_time) / 1000000000;


  printf("GPU Vector: Time %1.2f s\n", gpu_time);
  // Query the s-sparse sampler and print out
  // size   = 0;
  // result = query(gpu_sampler, size);

  // for (i = 0; i < size; i++) printf("%u ", result[i]);
  // printf("\n");

  // Clean up
  hipFree(buffer);
  // hipFree(buffer2);

  // hipFree(d_samplers);
}

int main(int argc, char **argv) {
  // Reads datafile from args
  if(argc < 2){
    printf("Usage: %s <input_file>", argv[0]);
    return 0;
  }
  sample(argv[1], 25, 50, 20);
  return 0;
}
