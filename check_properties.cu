#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int main(int argc, char *argv[]){
  struct hipDeviceProp_t properties;
  hipGetDeviceProperties(&properties, 0);
  printf("using %i multiprocessors\n max threads per processor: %i \n"
    ,properties.multiProcessorCount
    ,properties.maxThreadsPerMultiProcessor);
  return 0;
}
